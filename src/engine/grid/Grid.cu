#include "hip/hip_runtime.h"
#include <engine/grid/Grid.cuh>

__host__ Grid::Grid(std::vector<Sphere>& spheres) : _totalNumSpheres(spheres.size()), _h_Spheres(spheres), _d_Spheres(_h_Spheres)
{
    ComputeGridSize();
    ComputeGridResolution();
    Populate();
    CopyCellsToDevice();
}

// Intersect traverses the grid and checks for sphere hits using 3D-DDA algorithm.
__device__ bool Grid::Intersect(const Ray& ray, HitData& hit)
{
    glm::vec3 normalisedRayDir = glm::normalize(ray.direction);

    // Check if the ray intersects the grid using AABB test.
    float tGridEntry = 0.00001f, tGridExit = INFINITY;
    for (int i = 0; i < 3; ++i) 
    {
        float tMin = (_gridMin[i] - ray.origin[i]) / normalisedRayDir[i];
        float tMax = (_gridMax[i] - ray.origin[i]) / normalisedRayDir[i];
        // Make sure tMin is always smaller than tMax.
        if (tMin > tMax) 
        {
            float tempTMin = tMin;
            tMin = tMax;
            tMax = tempTMin;
        }

        // Also set a valid hit interval for the ray to the grid boundaries.
        tGridEntry = glm::max(tMin, tGridEntry);
        tGridExit = glm::min(tMax, tGridExit);
        if (tGridEntry > tGridExit) 
        { 
            return false;
        }
    }

    // Convert ray's origin to the cell coordinates.
    glm::vec3 gridRelativeRayOrigin = ray.origin - _gridMin;
    glm::ivec3 cell = GetCellCoords(ray.origin);

    // Calculate initial t, deltaT and step values.
    glm::vec3 deltaT = glm::vec3(0.0f), t = glm::vec3(0.0f);
    glm::ivec3 step = glm::ivec3(0);
    for (int i = 0; i < 3; ++i) 
    { 
        if (normalisedRayDir[i] > 0) // Positive ray direction
        {
            t[i] = ((cell[i] + 1) * _cellSize[i] - gridRelativeRayOrigin[i]) / normalisedRayDir[i]; // Add '1' to cell to get next boundary index.
            deltaT[i] = _cellSize[i] / normalisedRayDir[i];
            step[i] = 1;
        }
        else
        {
            t[i] = (cell[i] * _cellSize[i] - gridRelativeRayOrigin[i]) / normalisedRayDir[i];
            deltaT[i] = -(_cellSize[i] / normalisedRayDir[i]); // Makes sure deltaT is always positive for accurate traversing.
            step[i] = -1;
        }
    }

    // Traverse.
    while (true) {
        const int cellIdx = GetCellIndex(cell.x, cell.y, cell.z);
        const Cell& currentCell = _d_Cells[cellIdx];
        const Sphere* spheres = thrust::raw_pointer_cast(_d_Spheres.data());
        const int numSpheres = currentCell.GetNumSpheres();
        
        if (currentCell.Intersect(spheres, numSpheres, ray, tGridEntry, tGridExit, hit)) // Using grid boundaries for hit interval.
        {
            return true;
        }

        // Step to the next cell depending on the smallest intersection point.
        if (t.x < t.y && t.x < t.z) 
        {
            t.x += deltaT.x;
            cell.x += step.x;
        }
        else if (t.y < t.z) 
        {
            t.y += deltaT.y;
            cell.y += step.y;
        }
        else 
        {
            t.z += deltaT.z;
            cell.z += step.z;
        }

        // Break when the ray is out of bounds.
        if (
            cell.x < 0 || cell.x >= _gridResolution.x ||
            cell.y < 0 || cell.y >= _gridResolution.y ||
            cell.z < 0 || cell.z >= _gridResolution.z
        ) 
        {
            break;
        }
    }

    return false;
}

// ComputeGridSize calcualtes the sum of bboxes of all spheres which is the grid size.
__host__ void Grid::ComputeGridSize()
{
    _gridMin = _h_Spheres[0].GetCenter() - _h_Spheres[0].GetRadius();
    _gridMax = _h_Spheres[0].GetCenter() + _h_Spheres[0].GetRadius();
    for (const Sphere& sphere : _h_Spheres)
    {
        glm::vec3 sphereMin = sphere.GetCenter() - sphere.GetRadius();
        glm::vec3 sphereMax = sphere.GetCenter() + sphere.GetRadius();
        _gridMin = glm::min(_gridMin, sphereMin);
        _gridMax = glm::max(_gridMax, sphereMax);
    }
    _gridSize = _gridMax - _gridMin;

    printf("Grid Min: (%f, %f, %f) \n", _gridMin.x, _gridMin.y, _gridMin.z);
    printf("Grid Max: (%f, %f, %f) \n", _gridMax.x, _gridMax.y, _gridMax.z);
    printf("Grid Size: (%f, %f, %f) \n", _gridSize.x, _gridSize.y, _gridSize.z);
}

// ComputeGridResolution computes grid resolution based on the number of spheres and the scene overall volume.
__host__ void Grid::ComputeGridResolution()
{
    float volume = _gridSize.x * _gridSize.y * _gridSize.z;
    float cubeRoot = std::powf( _totalNumSpheres / volume, 1.0f / 3.0f);
  
    _gridResolution = glm::max(glm::floor(_gridSize * cubeRoot), glm::vec3(1)); // Make sure it is atleast 1.
    _cellSize = _gridSize / _gridResolution;

    printf("Grid Resolution: (%f, %f, %f) \n", _gridResolution.x, _gridResolution.y, _gridResolution.z);
    printf("Cell Size: (%f, %f, %f) \n", _cellSize.x, _cellSize.y, _cellSize.z);
}

// Populate populates the grid cells with sphere indexes.
__host__ void Grid::Populate()
{
    _cellSize = _gridSize / _gridResolution;
    int numOfCells = _gridResolution.x * _gridResolution.y * _gridResolution.z;
    _h_Cells.resize(numOfCells);
    printf("Number of cells: %d \n", _h_Cells.size());

    for (int sphereIdx = 0; sphereIdx < _h_Spheres.size(); ++sphereIdx)
    {
        const Sphere& sphere = _h_Spheres[sphereIdx];
        glm::vec3 sphereBBoxMin = sphere.GetCenter() - sphere.GetRadius();
        glm::vec3 sphereBBoxMax = sphere.GetCenter() + sphere.GetRadius();

        // Convert to cell coords.
        glm::ivec3 minCell = glm::floor((sphereBBoxMin - _gridMin) / _cellSize);
        glm::ivec3 maxCell = glm::floor((sphereBBoxMax - _gridMin) / _cellSize);

        // Clamp to make sure we are within the grid's boundaries.
        minCell = glm::clamp(minCell, glm::ivec3(0), glm::ivec3(_gridResolution - 1.0f));
        maxCell = glm::clamp(maxCell, glm::ivec3(0), glm::ivec3(_gridResolution - 1.0f));

        // Insert sphere indexes.
        for (int z = minCell.z; z <= maxCell.z; ++z)
        {
            for (int y = minCell.y; y <= maxCell.y; ++y)
            {
                for (int x = minCell.x; x <= maxCell.x; ++x)
                {
                    int cellIdx = GetCellIndex(x,y,z);
                    printf("Adding Sphere index '%d' to Cell at index '%d'\n", sphereIdx, cellIdx);
                    _h_Cells[cellIdx].Add(sphereIdx);
                }
            }
        }
    }
}

// CopyCellsToDevice allocates and copies cells array and internal cell data to the device.
__host__ void Grid::CopyCellsToDevice()
{
    size_t numCells = _h_Cells.size();
    CUDA_CHECK_ERROR(hipMalloc(&_d_Cells, numCells * sizeof(Cell)));
    for (int i = 0; i < numCells; ++i)
    {
        // Internal cell data must be copied to the device as well.
        _h_Cells[i].AllocateDeviceMemory();
        _h_Cells[i].CopyToDevice();
        CUDA_CHECK_ERROR(hipMemcpy(&_d_Cells[i], &_h_Cells[i], sizeof(Cell), hipMemcpyHostToDevice));
    }
}

// GetCellCoords gets cell coordinates relative to the worldPos.
__device__ glm::vec3 Grid::GetCellCoords(const glm::vec3& worldPos) const
{
    glm::vec3 gridRelativeCoords = (worldPos - _gridMin) / _cellSize;
    // Floor to get the starting cell boundary index.
    glm::vec3 lowerCellIndx = glm::floor(gridRelativeCoords);
    // Clamp to make sure the cell is within grid's boundaries.
    return glm::clamp(lowerCellIndx, glm::vec3(0.0f), _gridResolution - 1.0f);
}

// GetCellIndex convert's 3D coordinates to 1D index.
__device__ __host__ int Grid::GetCellIndex(int x, int y, int z) const
{
    return x + y * _gridResolution.x + z * _gridResolution.x * _gridResolution.y;
}