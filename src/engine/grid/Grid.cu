#include "hip/hip_runtime.h"
#include <engine/grid/Grid.cuh>

#define CUDA_CHECK_ERROR(call)                                          \
    do {                                                                \
        hipError_t err = call;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << "CUDA error " << err << " at " << __FILE__ <<  \
            ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                         \
        }                                                               \
    } while (0)

__host__ Grid::Grid(std::vector<Sphere>& spheres)
{
    _numSpheres = spheres.size();
    _h_Spheres = spheres;
    _d_Spheres = _h_Spheres;
    ComputeSceneBoundingBox();
    ComputeGridResolution();
    // Populate cells:
    _cellSize = _gridSize / _gridResolution;
    int totalCells = _gridResolution.x * _gridResolution.y * _gridResolution.z;
    _h_Cells.resize(totalCells);
    

    Populate();
    // Copy to the GPU:
    CopyCellsToDevice();
    std::cout << "Finish setting up Grid" << std::endl;
}

__host__ Grid::~Grid() 
{

}

__device__ bool Grid::Intersect(const Ray& ray, float tMin, float tMax, HitData& hit)
{
    glm::vec3 rayGridOrigin = ray.origin - _gridMin;
    glm::vec3 originCell = rayGridOrigin / _cellSize;
    glm::vec3 normalisedRayDir = glm::normalize(ray.direction);
    glm::vec3 deltaT = glm::vec3(0), t = glm::vec3(0);

    // AABB (Axis-Aligned Bounding Box) intersection test
    //float t0 = tMin, t1 = tMax;
    //for (int i = 0; i < 3; ++i)
    //{
    //    float invDir = 1.0f / normalisedRayDir[i];
    //    float tNear = (_gridMin[i] - ray.origin[i]) * invDir;
    //    float tFar = (_gridMax[i] - ray.origin[i]) * invDir;

    //    if (tNear > tFar)
    //    {
    //        // Swap
    //        float temp = tNear;
    //        tNear = tFar;
    //        tFar = temp;
    //    }

    //    t0 = tNear > t0 ? tNear : t0;
    //    t1 = tFar < t1 ? tFar : t1;

    //    if (t0 > t1) 
    //    {
    //        return false; 
    //    }
    //}
    for (int i = 0; i < 3; ++i)
    {
        if (normalisedRayDir[i] > 0) // Positive direction.
        {
            t[i] = ((floor(originCell[i]) + 1) * _cellSize[i] - rayGridOrigin[i]) / normalisedRayDir[i];
        }
        else // Negative direction.
        {
            t[i] = ((ceil(originCell[i]) - 1) * _cellSize[i] - rayGridOrigin[i]) / normalisedRayDir[i];
        }
        deltaT[i] = _cellSize[i] / std::abs(normalisedRayDir[i]);
    }

    float currentT = 0.0f;
    while (1)
    {
        //printf("Called");
        // Check if the ray intersects any spheres in the current cell
        //int cellIdx = GetCellIndex(static_cast<int>(originCell.x), static_cast<int>(originCell.y), static_cast<int>(originCell.z));
        //printf("Cell index: %d" + cellIdx + '\n');
        if (_d_Cells[0].Intersect(thrust::raw_pointer_cast(_d_Spheres.data()), _d_Cells[0]._h_NumSpheres, ray, tMin, tMax, hit))
        {
            tMax = hit.t;
            return true;
        }
        

        // Determine the next cell to step to:
        if (t.x < t.y)
        {
            currentT = t.x;
            t.x += deltaT.x;
            if (normalisedRayDir.x > 0) // Positive direction.
            {
                originCell.x += 1;
            }
            else // Negatve direction.
            {
                originCell.x -= 1;
            }
        }
        else if (t.y < t.z)
        {
            currentT = t.y;
            t.y += deltaT.y;
            if (normalisedRayDir.y > 0) // Positive direction.
            {
                originCell.y += 1;
            }
            else // Negatve direction.
            {
                originCell.y -= 1;
            }
        }
        else
        {
            currentT = t.z;
            t.z += deltaT.z;
            if (normalisedRayDir.z > 0) // Positive direction.
            {
                originCell.z += 1;
            }
            else // Negatve direction.
            {
                originCell.z -= 1;
            }
        }

        if (originCell.x < 0 || originCell.x > _gridMax.x - 1.0f ||
            originCell.y < 0 || originCell.y > _gridMax.y - 1.0f ||
            originCell.z < 0 || originCell.z > _gridMax.z - 1.0f)
        {
            break;
        }
    }

    return false;
}

__host__ void Grid::ComputeSceneBoundingBox()
{
    _gridMin = _h_Spheres[0].GetCenter() - _h_Spheres[0].GetRadius();
    _gridMax = _h_Spheres[0].GetCenter() + _h_Spheres[0].GetRadius();
    for (const Sphere& sphere : _h_Spheres)
    {
        glm::vec3 sphereMin = sphere.GetCenter() - sphere.GetRadius();
        glm::vec3 sphereMax = sphere.GetCenter() + sphere.GetRadius();
        _gridMin = glm::min(_gridMin, sphereMin);
        _gridMax = glm::min(_gridMax, sphereMax);
    }
    _gridSize = _gridMax - _gridMin;
}

__host__ void Grid::ComputeGridResolution()
{
    int numOfSpheres = _h_Spheres.size();
    float volume = _gridSize.x * _gridSize.y * _gridSize.z;
    float cubeRoot = std::pow(lambda * numOfSpheres / volume, 1 / 3);

    _gridResolution = glm::vec3(_gridSize * cubeRoot);
}

__host__ void Grid::Populate()
{
    for (int i = 0; i < _h_Spheres.size(); ++i)
    {
        const Sphere& sphere = _h_Spheres[i];
        glm::vec3 sphereBBoxMin = sphere.GetCenter() - sphere.GetRadius();
        glm::vec3 sphereBBoxMax = sphere.GetCenter() + sphere.GetRadius();

        glm::vec3 minCell = glm::floor(sphereBBoxMin / _cellSize);
        glm::vec3 maxCell = glm::ceil(sphereBBoxMax / _cellSize);

        for (int z = minCell.z; z <= maxCell.z; ++z)
        {
            for (int y = minCell.y; y <= maxCell.y; ++y)
            {
                for (int x = minCell.x; x <= maxCell.x; ++x)
                {
                    if (x >= 0 && x < _gridResolution.x &&
                        y >= 0 && y < _gridResolution.y &&
                        z >= 0 && z < _gridResolution.z)
                    {
                        int cellIdx =GetCellIndex(x,y,z);
                        std::cout << "Adding sphereIdx: " << i << " to Cell at index: " << cellIdx << std::endl;
                        _h_Cells[cellIdx].Add(i);
                    }
                }
            }
        }
    }
}

__host__ void Grid::CopyCellsToDevice()
{
    // Allocate memory for cells on the device
    size_t numCells = _h_Cells.size();
    CUDA_CHECK_ERROR(hipMalloc((void**)&_d_Cells, numCells * sizeof(Cell)));

    // Allocate memory for each cell's internal device data
    for (int i = 0; i < numCells; ++i)
    {
        std::cout << "Allocating device memory for cell " << i << std::endl;
        _h_Cells[i].AllocateDeviceMemory();
    }

    // Copy each cell from host to device
    for (int i = 0; i < numCells; ++i)
    {
        std::cout << "Copying cell " << i << " with " << _h_Cells[i]._h_NumSpheres << " spheres to device." << std::endl;
        _h_Cells[i].CopyToDevice();
        CUDA_CHECK_ERROR(hipMemcpy(&_d_Cells[i], &_h_Cells[i], sizeof(Cell), hipMemcpyHostToDevice));
    }

    CUDA_CHECK_ERROR(hipDeviceSynchronize());
}

__host__ glm::vec3 Grid::GetCellCoords(const glm::vec3& worldPos) const
{
    glm::vec3 coords = (worldPos - _gridMin) / _cellSize;
    return glm::clamp(coords, glm::vec3(0.0f), _gridResolution - 1.0f);

}

__device__ __host__ int Grid::GetCellIndex(int x, int y, int z) const
{
    // Convert 3D coords to 1D index using x + y * width + z * width * height.
    return x + y * _gridResolution.x + z * _gridResolution.x * _gridResolution.y;
}