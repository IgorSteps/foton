#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <engine/Renderer.h>



__device__ Ray GetRay(const CameraData* cam, float u, float v) {
    float tanFovHalf = tanf(glm::radians(cam->fov / 2.0f));

    float ndcX = (2.0f * u) - 1.0f;
    float ndcY = 1.0f - (2.0f * v);

    float camX = ndcX * cam->aspectRatio * tanFovHalf;
    float camY = ndcY * tanFovHalf;

    glm::vec3 rayDirection = glm::normalize(cam->front + camX * cam->right - camY * cam->up);
    return Ray{ cam->position, rayDirection };
}

__device__ bool isInShadow(const Ray& ray, const Sphere* d_spheres, const int numOfSpheres, float lightDist)
{
    HitData tempHit;
    for (int i = 0; i < numOfSpheres; ++i)
    {
        if (d_spheres[i].Hit(ray, 0.001f, lightDist, tempHit))
        {
            if (!d_spheres[i].IsLight())
            {
                return true;
            }
            else
            {
                return false;
            }
        }
    }

    return false;
}

__device__ glm::vec3 ComputePhongIllumination(
    Light* light,
    const HitData& hit,
    const Sphere* d_spheres,
    int numOfSpheres,
    const glm::vec3& objectColor
)
{
    glm::vec3 lightDir = glm::normalize(light->position - hit.point);

    // Setup shadow ray.
    Ray shadowRay;

    shadowRay.origin = hit.point;
    shadowRay.direction = lightDir;
    float distanceToLight = glm::length(light->position - hit.point);

    // Check if the point is in shadow
    bool inShadow = isInShadow(shadowRay, d_spheres, numOfSpheres, distanceToLight);

    // Ambient.
    float ambientStrength = 0.1f;
    glm::vec3 ambient = ambientStrength * light->color;

    if (inShadow) 
    {
        // If in shadow, only ambient light
        return ambient * objectColor;
    }
    else 
    {
        // Diffuse.
        float diff = max(glm::dot(hit.normal, lightDir), 0.0f);
        glm::vec3 diffuse = diff * light->color;

        glm::vec3 result = (diffuse + ambient) * objectColor;
        return result;
    }
}


__global__
void renderKernel(glm::vec3* output, int width, int height, CameraData* camData, Sphere* d_spheres, int numOfSpheres, Light* d_light, Ground* d_ground)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    // Normalise screen coordinates
    float u = float(i) / (width - 1);
    float v = float(j) / (height - 1);

    Ray ray = GetRay(camData, u, v);
    HitData hitData;

    // Keeps track of the closest hit.
    float closestSoFar = INFINITY;
    // Keeps if we've hit anything.
    bool hitSomething = false;
    // Start with black colour.
    glm::vec3 color = glm::vec3(0.0f);

    for (int x = 0; x < numOfSpheres; x++)
    {
        if (d_spheres[x].Hit(ray, 0.001f, closestSoFar, hitData))
        {
            closestSoFar = hitData.t;
            hitSomething = true;

            if (!d_spheres[x].IsLight())
            {
                color += ComputePhongIllumination(d_light, hitData, d_spheres, numOfSpheres, d_spheres[x].GetColour());
            }
            else
            {
                color = d_spheres[x].GetColour();
            }
        }
    }

    if (!hitSomething && d_ground->Hit(ray, closestSoFar, hitData))
    {
        closestSoFar = hitData.t;
        hitSomething = true;
        color = ComputePhongIllumination(d_light, hitData, d_spheres, numOfSpheres, d_ground->groundColor);
    }

    if (!hitSomething)
    {
        glm::vec3 unitDirection = glm::normalize(ray.direction);
        auto a = 0.5f * (unitDirection.y + 1.0f);
        color = (1.0f - a) * glm::vec3(1.0f, 1.0f, 1.0f) + a * glm::vec3(0.5f, 0.7f, 1.0f);
    }


    output[j * width + i] = color;
}



// For debugging
// 
__global__ void printDebugSphereProperties(Sphere* spheres, int numSpheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int x = 0; x < numSpheres; ++x)
        {
            // Print properties of the first sphere
            printf("Sphere %i - Center: (%f, %f, %f), Radius: %f\n",
                x,
                spheres[x].GetCenter().x,
                spheres[x].GetCenter().y,
                spheres[x].GetCenter().z,
                spheres[x].GetRadius()
            );
        }
    }
}

void Renderer::RenderUsingCUDA(void* cudaPtr, int numOfSphere)
{
    // Launch CUDA kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(
        (1200 + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (800 + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // TODO: Get width/height from engine
    renderKernel << <numBlocks, threadsPerBlock >> > (static_cast<glm::vec3*>(cudaPtr), 1200, 800, d_cameraData, d_spheres, numOfSphere, d_light, d_Ground);
    //printDebugSphereProperties << <1, 1 >> > (d_spheres, numOfSphere);

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error in kernel launch: %s\n", hipGetErrorString(error));
    }
}